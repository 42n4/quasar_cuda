#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_tools.cc"

#define THREADS_NUM 512

__global__ void matrix_log10
	(
		double *matrix,	// Macierz
		uint row_size,			// Rozmiar wiersza macierzy.
		uint col_size
	)
{
	// gid0 - numer wiersza macierzy input
	uint gid0 = blockIdx.x * blockDim.x + threadIdx.x;
	// gid1 - numer elementu w wierszu.
	uint gid1 = blockIdx.y * blockDim.y + threadIdx.y;
	if(gid1 >= row_size || gid0 >= col_size)
	{
		return;
	}
	
	uint idx = gid0 * row_size + gid1;
	double m = matrix[idx];
	m = log10(m);
	matrix[idx] = m;
}

__global__
void matrix_add_scalar
	(
		double * matrix,
		uint row_size, //TODO: change to width height
		uint col_size,
		const double scalar
	)
{
	// gid0 - numer wiersza macierzy input
	const uint gid0 = blockIdx.x * blockDim.x + threadIdx.x;
	// gid1 - numer elementu w wierszu.
	const uint gid1 = blockIdx.y * blockDim.y + threadIdx.y;

	if(gid1 >= row_size || gid0 >= col_size)
		return;
	
	const uint idx = gid0 * row_size + gid1;
	matrix[idx] = matrix[idx] + scalar;
}

__global__
void matrix_minus_matrix
	(
		const double* minuend,	// Macierz
					// Rozmiar wiersza macierzy.
		double* subtrahend,		// Macierz do odjęcia
		double* output,	// Wynik
		const uint row_size,
		const uint col_size
	)
{
	// gid0 - numer wiersza macierzy input
	uint gid0 = blockIdx.x * blockDim.x + threadIdx.x;
	// gid1 - numer elementu w wierszu.
	uint gid1 = blockIdx.y * blockDim.y + threadIdx.y;

	if(gid1 >= row_size || gid0 >= col_size)
	{
		return;
	}
	
	const uint idx = gid0 * row_size + gid1;
	output[idx] = minuend[idx] - subtrahend[idx];
}

__global__
void matrix_divide_matrix
	(
		const double* divident,	// Macierz
					// Rozmiar wiersza macierzy.
		const double* divisor,		// Macierz do odjęcia
		double* output,	// Wynik
		const uint row_size,
		const uint col_size
	)
{
	// gid0 - numer wiersza macierzy input
	uint gid0 = blockIdx.x * blockDim.x + threadIdx.x;
	// gid1 - numer elementu w wierszu.
	uint gid1 = blockIdx.y * blockDim.y + threadIdx.y;

	if(gid1 >= row_size || gid0 >= col_size)
	{
		return;
	}
	
	const uint idx = gid0 * row_size + gid1;
	output[idx] = divident[idx] / divisor[idx];
}

__global__
void matrix_multiply_vector
	(
		const double *matrix,
		const double *vector,	// Wektor, których zawiera co najmniej
		double *output,	// Wynik
		const uint row_size,
		const uint col_size
	)
{
	// gid0 - numer wiersza macierzy input
	uint gid0 = blockIdx.x * blockDim.x + threadIdx.x;
	// gid1 - numer elementu w wierszu (numer kolumny).
	uint gid1 = blockIdx.y * blockDim.y + threadIdx.y;

	if(gid1 >= row_size || gid0 >= col_size)
		return;
	
	uint idx = gid0 * row_size + gid1;
	uint col_idx = gid1;
	
	output[idx] = matrix[idx] * vector[col_idx];
}

__global__
void matrix_transpose(
  double *input,
  double *output,
  const uint row_size,
  const uint col_size
)
{

}


extern "C"
void matrixAddScalar(double* h_input, 
               const size_t width, 
               const size_t height,
               const double scalar
) {
  // initialize variables
  double *d_input;
  size_t size = width * height * sizeof(double);
  hipError_t cudaStatus;
  // initialize device
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
      return;
  }
  // kernel memory allocation
  cudaStatus = hipMalloc((void**)&d_input, size);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed!");
      return;
  }
  
  cudaStatus = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      return;
  }

  // run kernel
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid(1, 1);
  blocksPerGrid.x = ceil(double(width)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(height)/double(threadsPerBlock.y));
  matrix_add_scalar<<<blocksPerGrid, threadsPerBlock>>>(
    d_input,
    width,
    height,
    scalar
  );
  
  // device to host memory copy
  cudaStatus = hipMemcpy(h_input, d_input, size, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
  }
  // free memory
  hipFree(d_input);
}

extern "C"
void matrixLog10(double* h_input, const size_t width, size_t height)
{
    double *d_input = 0;
    size_t size = width * height * sizeof(double);
    //initialize device 
    checkCudaErrors(hipSetDevice(0));
  
    // device memory allocation
    checkCudaErrors(hipMalloc((void**)&d_input, size));
    // device memory copying
    checkCudaErrors(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
    //kernel invocation
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(1, 1);
    blocksPerGrid.x = ceil(double(width)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(height)/double(threadsPerBlock.y));
    matrix_log10<<<blocksPerGrid, threadsPerBlock>>>(d_input, width, height);
    checkCudaErrors(hipGetLastError());
    //device to host memory copy
    checkCudaErrors(hipMemcpy(h_input, d_input, size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
}

extern "C"
void matrixMultiplyColVector(
    const double* h_input,
    double* h_output,
    const double* h_vector,
    const size_t width,
    const size_t height,
    const size_t length
) 
{
  double *d_input = 0, *d_output = 0, *d_vector = 0;
  const size_t size = width * height * sizeof(double);
  const size_t vSize = length * sizeof(double);
  
  checkCudaErrors(hipSetDevice(0));
  checkCudaErrors(hipMalloc((void**)&d_input, size));
  checkCudaErrors(hipMalloc((void**)&d_output, size));
  checkCudaErrors(hipMalloc((void**)&d_vector, vSize)); //TODO: check!
  checkCudaErrors(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_vector, h_vector, vSize, hipMemcpyHostToDevice));
  //kernel invocation
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid(1, 1);
  blocksPerGrid.x = ceil(double(width)/double(threadsPerBlock.x));
  blocksPerGrid.y = ceil(double(height)/double(threadsPerBlock.y));
  matrix_multiply_vector<<<blocksPerGrid, threadsPerBlock>>>(
    d_input,
    d_vector, 
    d_output, 
    width, 
    height
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost));
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_vector);
}

extern "C"
void matrixSubstractMatrix(
  const double* h_input,
  const double* h_subtrahend,
  double* h_output,
  const size_t width,
  const size_t height
) 
{
   double* d_input = 0, *d_subtrahend = 0, *d_output = 0;
    size_t size = width * height * sizeof(double);
    //initialize device 
    checkCudaErrors(hipSetDevice(0));
    
    // device memory allocation
    checkCudaErrors(hipMalloc((void**)&d_input, size));
    
    checkCudaErrors(hipMalloc((void**)&d_subtrahend, size));
    checkCudaErrors(hipMalloc((void**)&d_output, size));
    
    // device memory copying
    checkCudaErrors(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_subtrahend, h_subtrahend, size, hipMemcpyHostToDevice));
    //kernel invocation
    dim3 threadsPerBlock(512, 2);
    dim3 blocksPerGrid(1, 1);
    blocksPerGrid.x = ceil(double(width)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(height)/double(threadsPerBlock.y));
    matrix_minus_matrix<<<blocksPerGrid, threadsPerBlock>>>(
      d_input,
      d_subtrahend,
      d_output,
      width, 
      height
    );
    //device to host memory copy
    checkCudaErrors(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost));
    hipFree(d_input);
    hipFree(d_subtrahend);
    hipFree(d_output);
}


extern "C"
void matrixDivideMatrix(
    double* h_divided,
    double* h_divisor,
    double* h_output,
    const size_t width,
    const size_t height
)
{
   double* d_divided = 0, *d_divisor = 0, *d_output = 0;
    size_t size = width * height * sizeof(double);
    //initialize device 
    checkCudaErrors(hipSetDevice(0));
    
    // device memory allocation
    checkCudaErrors(hipMalloc((void**)&d_divided, size));
    
    checkCudaErrors(hipMalloc((void**)&d_divisor, size));
    checkCudaErrors(hipMalloc((void**)&d_output, size));
    
    // device memory copying
    checkCudaErrors(hipMemcpy(d_divided, h_divided, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_divisor, h_divisor, size, hipMemcpyHostToDevice));
    //kernel invocation
    dim3 threadsPerBlock(512, 2);
    dim3 blocksPerGrid(1, 1);
    blocksPerGrid.x = ceil(double(width)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(height)/double(threadsPerBlock.y));
    matrix_divide_matrix<<<blocksPerGrid, threadsPerBlock>>>(
      d_divided,
      d_divisor,
      d_output,
      width, 
      height
    );
    //device to host memory copy
    checkCudaErrors(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost));
    hipFree(d_divided);
    hipFree(d_divisor);
    hipFree(d_output);
}



extern "C"
void matrixTranspose(
    double *h_input,
    double *h_output,
    const size_t realWidth,
    const size_t realHeight
) 
{
    double *d_input = 0, *d_output = 0;
    // for efficient kernel computations, we should expand matrices
    
    size_t size = realWidth * realHeight * sizeof(double);
    //initialize device 
    checkCudaErrors(hipSetDevice(0));
  
    // device memory allocation
    checkCudaErrors(hipMalloc((void**)&d_input, size));
    checkCudaErrors(hipMalloc((void**)&d_output, size));
    // device memory copying
    checkCudaErrors(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));
    //kernel invocation
    dim3 threadsPerBlock(512, 2);
    dim3 blocksPerGrid(1, 1);
    blocksPerGrid.x = ceil(double(realWidth)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(realHeight)/double(threadsPerBlock.y));
    matrix_transpose<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, realWidth, realHeight);
    checkCudaErrors(hipGetLastError());
    
    //device to host memory copy
    checkCudaErrors(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost));
    
    hipFree(d_input);
    hipFree(d_output);
}